#include "hip/hip_runtime.h"
/*
	STUDENT NAME: Gökay Gülsoy
	STUDENT NO: 270201072
*/

#include <math.h>
#include <stdio.h>
#include <stdlib.h>
#include "timer.h"
#include "files.h"

#define SOFTENING 1e-9f

/*
 * Each body contains x, y, and z coordinate positions,
 * as well as velocities in the x, y, and z directions.
 */

typedef struct { float x, y, z, vx, vy, vz; } Body;

/*
   bodyForce kernel that calculates the gravitational
   impact of all bodies in the system on all others
*/

__global__ void bodyForce(Body* p,float dt,int n) {
        
        int i = threadIdx.x + blockDim.x * blockIdx.x;
        
        if (i < n) {
           float Fx = 0.0f; float Fy = 0.0f; float Fz = 0.0f;   
           
            for (int j = 0; j < n; j++) {
                float dx = p[j].x - p[i].x;    
                float dy = p[j].y - p[i].y;
                float dz = p[j].z - p[i].z;
                
                float distSqr = dx*dx + dy*dy + dz*dz + SOFTENING;
                float invDist = rsqrtf(distSqr);
                float invDist3 = invDist * invDist * invDist;
                
                Fx += dx * invDist3; Fy = dy * invDist3; Fz += dz * invDist3;
            }
            
            p[i].vx += dt * Fx; p[i].vy += dt * Fy; p[i].vz += dt * Fz;
        }
}


int main(const int argc, const char** argv) {

  // The assessment will test against both 2<11 and 2<15.
  // Feel free to pass the command line argument 15 when you generate ./nbody report files
  int nBodies = 2<<11;
  if (argc > 1) nBodies = 2<<atoi(argv[1]);

  // The assessment will pass hidden initialized values to check for correctness.
  // You should not make changes to these files, or else the assessment will not work.
  const char * initialized_values;
  const char * solution_values;

  if (nBodies == 2<<11) {
    initialized_values = "09-nbody/files/initialized_4096";
    solution_values = "09-nbody/files/solution_4096";
  } else { // nBodies == 2<<15
    initialized_values = "09-nbody/files/initialized_65536";
    solution_values = "09-nbody/files/solution_65536";
  }

  if (argc > 2) initialized_values = argv[2];
  if (argc > 3) solution_values = argv[3];

  const float dt = 0.01f; // Time step
  const int nIters = 10;  // Simulation iterations

  float* dt, int* n;
  float* dt_device, int* n_device;
    
  int bytes = nBodies * sizeof(Body);
  float *buf;

  buf = (float *)malloc(bytes);

  Body *p = (Body*)buf;

  Body* p_device;
  
  dt = (float *)malloc(sizeof(float));          
  n = (int *)malloc(sizeof(int));
    
  *dt = 0.01f;  
  *n = nBodies;  
 
  read_values_from_file(initialized_values, buf, bytes);
  Body* p_device;
    
  // allocating memory on device  
  hipError_t p_err = hipMalloc((void **)&p_device,bytes);
  hipError_t dt_err = hipMalloc((void **)&dt_device,sizeof(float));
  hipError_t n_err = hipMalloc((void **)&n_device,sizeof(int));  
  
  // error checks for malloc functions
  if (p_err != hipSuccess) {
     printf("%s error has occured",hipGetErrorString(p_err));
     exit(EXIT_FAILURE);
  }

  if (dt_err != hipSuccess) {
     printf("%s error has occured",hipGetErrorString(dt_err));
     exit(EXIT_FAILURE);
  }    
   
  if (n_err != hipSuccess) {
     printf("%s error has occured",hipGetErrorString(n_err));
     exit(EXIT_FAILURE);
  }    
    
  double totalTime = 0.0;

  // copying initial values from host to device  
  hipMemcpy(p_device,p,bytes,hipMemcpyHostToDevice);
  hipMemcpy(dt_device,dt,bytes,hipMemcpyHostToDevice);   
  hipMemcpy(n_device,n,bytes,hipMemcpyHostToDevice);
    
  /*
   * This simulation will run for 10 cycles of time, calculating gravitational
   * interaction amongst bodies, and adjusting their positions to reflect.
   */
  
  // calculating block and grid sizes  
  int blockSize = 256;
  int gridSize = (nBodies + blockSize -1) / blockSize;  
    
  for (int iter = 0; iter < nIters; iter++) {
    StartTimer();

    // launching cuda kernel   
    bodyForce<<<blockSize,gridSize>>>(p_device,dt_device,n_device); // compute interbody forces
    // waiting until all threads finish their job  
    hipDeviceSynchronize();  

    // copying modified values back to host
    hipMemcpy(p,p_device,bytes,hipMemcpyDeviceToHost);  
      
  /*
   * This position integration cannot occur until this round of `bodyForce` has completed.
   * Also, the next round of `bodyForce` cannot begin until the integration is complete.
   */
    
    for (int i = 0 ; i < nBodies; i++) { // integrate position
      p[i].x += p[i].vx*dt;
      p[i].y += p[i].vy*dt;
      p[i].z += p[i].vz*dt;
    }

    const double tElapsed = GetTimer() / 1000.0;
    totalTime += tElapsed;
  }

  double avgTime = totalTime / (double)(nIters);
  float billionsOfOpsPerSecond = 1e-9 * nBodies * nBodies / avgTime;
  write_values_to_file(solution_values, buf, bytes);

  // You will likely enjoy watching this value grow as you accelerate the application,
  // but beware that a failure to correctly synchronize the device might result in
  // unrealistically high values.
  printf("%0.3f Billion Interactions / second\n", billionsOfOpsPerSecond);

  // deallocating host and device memory  
  hipFree(p_device);
  hipFree(n_device);
  hipFree(dt_device);  
    
  free(p);
  free(n);
  free(dt);  
}
